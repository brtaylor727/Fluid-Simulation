#include "hip/hip_runtime.h"
#include <stdio.h>


//#define MATRIX_SIZE 64;
#define MATRIX_SIZE 16;


#define cudaErrorCheck(call) { cudaAssert(call,__FILE__,__LINE__); }

const double errortol = 1e-15;
const int numtimesteps = 1000;
const double beta = 0.0;

const double rho = 1e-3;
const double nu = 3e1;
const double g = 0e-5;


const double dt = 1e-13;
const double dx = 1e-4;

const double u0 = 1e-4;
const double phi0 = 0e2;

void cudaAssert(const hipError_t err, const char *file, const int line)
{ 
    if( hipSuccess != err) {                                                
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        
                file, line, hipGetErrorString(err) );
        exit(1);
    } 
}

__global__
void saxpy(int n, double a, double *x, double *y)
{
  int i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

__global__
//void jacobi_iterator_GPU(double  U_new[MATRIX_SIZE][MATRIX_SIZE], double U[MATRIX_SIZE][MATRIX_SIZE], double  F [MATRIX_SIZE][MATRIX_SIZE])
void jacobi_iterator_GPU(double * U_new, double* U, double * F ,int num)
{

    int N = MATRIX_SIZE;
    int j = blockDim.x * blockIdx.x + threadIdx.x ;
    int i = blockDim.y * blockIdx.y + threadIdx.y ;

    int ROW = blockIdx.y*blockDim.y+threadIdx.y;
    int COL = blockIdx.x*blockDim.x+threadIdx.x;

	//only apply to every other element
	if( (j*N+i) %2 != num){

		if ((i < N -1) && (j < N -1) && (i > 0) && (j > 0))
		{
			U_new[j * N + i] = (U[j * N + (i - 1)] + U[j * N + (i + 1)] + U[(j - 1) * N + i] + U[(j + 1) * N + i]  ) * 0.25 - (F[j*N+i]*.25);
		}
		/*
		else
		{


	if (i == 0 && (j !=0) && (j!=(N-1))){
	    //modified a minux one to a plus one
	    U_new[j*N+i] = ( 2*U[j * N + (i + 1)] + U[(j +1) * N + i] + U[(j -1) * N + i] ) * 0.25 + F[j*N+i];
	}
	else if ((i == (N-1))&&(j !=0) && (j!=(N-1))){
	   U_new[j*N+i] = (2*U[j * N + (i - 1)] + U[(j +1) * N + i] + U[(j -1) * N + i] ) * 0.25 + F[j*N+i];
	}
	else if ((j == 0) && (i!=0) && (i!=(N-1))){

	    U_new[j*N+i] = (2*U[(j+1) * N + i ] + U[j * N + i-1] + U[j * N + i+1] ) * 0.25 + F[j*N+i];
	}
	else if ((j == (N-1)) && (i!=0) && (i!=(N-1))){
	    U_new[j*N+i] = (2*U[(j-1) * N + i ] + U[j * N + i+1] + U[j * N + i-1] ) * 0.25 + F[j*N+i];
    	}
	else{
		if (i == j){
		U_new[j*N+i] = (U[(2*(j%(N-2))) * N + i ] + 2*U[j * N + (2*(i%(N-2))) ]) * 0.25 + F[j*N+i];
		}
	}
		

		}*/
	}else{
		U_new[j*N+i] = U[j*N+i];
	}


	    //boundary conditions (set the normal to zero)
	if ((i == 0 ))  {
		U_new[j*N+i] = U[j*N+i+1];
	}
	if ((i == (N-1) )){  
		U_new[j*N+i] = U[j*N+i-1];
	}
	if ((j == 0) && (i !=0) && (i!=(N-1))){ //don't apply to the corners let them be set by the y direction
		U_new[j*N+i] = U[(j+1)*N+i];
	}
	if ((j == (N-1)) &&  (i !=0) && (i!=(N-1))){
		U_new[j*N+i] = U[(j-1)*N+i];
	}

	//set the "ground"
	if ((i == 0)) {
	//	U_new[j*N+i] = 0;
	}//only need to set the pressure at the output?	
	if (i == (N-1)){
		
		//U_new[j*N+i] = dx*N*g;
		U_new[j*N+i] = phi0;
	}



    
}
__global__
void difference_GPU(double * U_new, double* U, double* diff_U)
{

	//compute the elementwise subtraction of two matrices
	//to be used for finding the maximum error in the poisson calculation

    int N = MATRIX_SIZE;
    int i = blockDim.x * blockIdx.x + threadIdx.x ;
    int j = blockDim.y * blockIdx.y + threadIdx.y ;

    //only access the internal elements of the array for now
    if ((i < N -1) && (j < N -1) && (i > 0) && (j > 0))
    {
       diff_U[j*N+i] = U_new[j * N + i] -  U[j*N+i];
    }
    else
    {
        diff_U[j*N + i] = 0.;
    }
}

double getmaxerror(double * A,double * B){
	double maxerror = 0.;
	int N = MATRIX_SIZE;
	for (int i = 0; i < N;i++){
		for (int j = 0; j < N;j++){
			double mydiff = abs(A[i*N+j] - B[i*N+j]);
			if (maxerror < mydiff){
				
		//		printf("maxerror: %f diffmat %f %i %i\n",maxerror,diffmat[i*N+j],i,j);
				maxerror = mydiff;
			}
		}
	}
	return maxerror;

}

void savemat(double * matrix,const char * name,int it,int N){

	char fn[100+1];
    
	snprintf(fn, 100, "./data/%s%04d.dat", name,it);

	FILE *f = fopen(fn, "w");
	if (f == NULL)
	{
		printf("Error opening file!\n");
		exit(1);
	}

	/* print integers and floats */
	for (int ix = 0;ix < N;ix++){
		for (int iy = 0; iy < N;iy++){
			fprintf(f, "%i %i %f\n", ix,iy,matrix[ix*N+iy]);
		}
	}
	
	fclose(f);
}


int main(void)
{

  int N = MATRIX_SIZE;




  //initialize velocity
  double *ux,*uy,*uintx,*uinty;
  ux = (double*)malloc(N*N*sizeof(double));
  uy = (double*)malloc(N*N*sizeof(double));
  uintx = (double*)malloc(N*N*sizeof(double));
  uinty = (double*)malloc(N*N*sizeof(double));

  for (int i = 0; i < N;i++){
	  for (int j = 0; j < N; j++){
		  
	  	ux[i*N+j] = 0.0;
	  	uy[i*N+j] = -.05*u0*(i-N/2)*(i-N/2)/N/N;
	  }
  }

  //initialize pressure
  double *p;
  p = (double*)malloc(N*N*sizeof(double));
  for (int i = 0; i < N*N;i++){
	  p[i] = phi0;
  }

  //initialize phi
  double *phi;
  phi = (double*)malloc(N*N*sizeof(double));
  for (int i = 0; i < N*N;i++){
	  phi[i] = phi0;
  }

  //initializae divu
  double *divu;
  divu = (double*)malloc(N*N*sizeof(double));
  for (int i = 0; i < N*N;i++){
	  divu[i] = 0.;
  }

  	//allocate memory for the U_new
	double *U_new,*U,*F,*diff_U;
	double *d_U_new,*d_U,*d_F,*d_diff_U;

	U_new = (double*)malloc(N*N*sizeof(double));
	U = (double*)malloc(N*N*sizeof(double));
	F = (double*)malloc(N*N*sizeof(double));
	diff_U = (double*)malloc(N*N*sizeof(double));

  	hipMalloc(&d_U_new,N*N*sizeof(double));
  	hipMalloc(&d_U,N*N*sizeof(double));
  	hipMalloc(&d_F,N*N*sizeof(double));




  	//TODO: add some method to add some numbers to my matrix

  	for (int i = 0; i < N*N;i++) {
		F[i] = 0.;
		U[i] = 4.0;
		U_new[i] = 5.0;
		diff_U[i] = 10.0;
	}




  //begin time iteration
  for(int it = 0;it < numtimesteps; it++){ //begin time iteration

	  //first step: calculate the intermediate velocity

	for (int ix = 1; ix < N-1;ix++){
		for (int iy = 1; iy < N-1;iy++){



			double ax = -dt*ux[ix*N+iy] * (ux[(ix+1)*N+iy]-ux[(ix-1)*N+iy])/(2*dx); 
			double bx = -beta*dt*(p[(ix+1)*N+iy]-p[(ix-1)*N+iy])/(2*dx*rho); 
			double cx = dt*nu*(ux[(ix-1)*N+iy] - 2*ux[ix*N+iy] + ux[(ix+1)*N+iy])/(dx*dx);
			double ddx = dt*0.0;
	

			
			uintx[ix*N+iy] = ux[ix*N+iy] + ax + bx + cx + ddx;
			//uintx[ix*N+iy] = 0.;


			double ay = -dt*uy[ix*N+iy] * (uy[ix*N+iy+1]-uy[ix*N+iy-1])/(2*dx); 
			double by = -beta*dt*(p[ix*N+iy+1]-p[ix*N+iy-1])/(2*dx*rho); 
			double cy = dt*nu*(uy[ix*N+iy-1] - 2*uy[ix*N+iy] + uy[ix*N+iy+1])/(dx*dx);
			double dy = -dt*g;

			//printf("ix: %d iy: %d ay: %.5e by: %.5e cy: %.5e dy: %.5e, uy: %.5e F: %.5e\n",ix,iy,ay,by,cy,dy,uy[ix*N+iy+1],F[ix*N+iy]);
			uinty[ix*N+iy] = uy[ix*N+iy] + ay + by + cy + dy;

			}
	}

			//boundary conditions for u_intermediate
	for (int ix = 0; ix < N;ix++){
		for (int iy = 0; iy < N;iy++){

			//set normals to left and right to be zero
			if ( (ix == 0)){
				//uinty[ix*N+iy] = uinty[(ix+1)*N+iy];
				uintx[ix*N+iy] = 0.;
				uinty[ix*N+iy] = 0.;
			}
			if ( (ix == (N-1))){
				//uinty[ix*N+iy] = uinty[(ix-1)*N+iy] ;
				uintx[ix*N+iy] = 0.;
				uinty[ix*N+iy] = 0.; 
			}
			//boundary condition for inlet
			if (iy ==(0)){

				//uinty[(N)*N+iy] = uinty[(N-1)*N+iy];
				uinty[(N)*ix+iy] = -u0;
				uintx[N*ix+iy] = 0.;
			}
			//boundary condition for the outlet 
			if(iy == (N-1)){
				uinty[N*ix+iy] = uinty[N*ix+iy-1];
			       uintx[N*ix+iy] = 0.;	
			
			}

			
//			printf("ix: %d iy: %d uinty: %.5e uintx: %.5e\n",ix,iy,uinty[ix*N+iy],uintx[ix*N+iy]);
		  }
	  }


	//second step: calculate the poisson equation to solve for phi


	//define F
	for (int ix = 1; ix < N-1;ix++){
		for (int iy = 1; iy < N-1;iy++){
			F[ix*N+iy] = .5*(dx/dt)*rho * ( uintx[(ix+1)*N+iy]- uintx[(ix-1)*N+iy]  
				     + uinty[ix*N+iy+1] - uinty[ix*N+iy-1]	);

			//printf("ix: %d iy: %d F: %.5e uinty: %.5e uintx: %.5e \n",ix,iy,F[ix*N+iy],uinty[ix*N+iy],uintx[ix*N+iy]);
		}

	}


	//boundary condition for F (first attempt)
	for (int ix = 0; ix < N;ix++){
		for (int iy = 0; iy < N;iy++){


			//boundary conditions for u_intermediate

			//set normals to left and right to be zero
			if ( (ix == 0)){

				//F[ix*N+iy] = F[(ix+1)*N+iy];
				F[ix*N+iy] = 0;
			}
			if ( (ix == (N-1))){
				//F[ix*N+iy] = F[(ix-1)*N+iy] ;

				F[ix*N+iy] = 0;
			}
			//boundary condition for inlet
			if (iy ==(0)){

				F[ix*N+iy] = 0;
				//F[(N)*N+iy] = uinty[(N-1)*N+iy];
				//F[(N)*ix+iy] = -u0;
			}
			//boundary condition for the outlet 
			if(iy == (N-1)){
				//F[N*ix+iy] = F[N*ix+iy-1];
			
				F[ix*N+iy] = 0;
			}

			
//			printf("ix: %d iy: %d uinty: %.5e uintx: %.5e\n",ix,iy,uinty[ix*N+iy],uintx[ix*N+iy]);
		  }
	  }

	

  	//send over the data to the gpu
	//hipMemcpy(d_U_new, U_new, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_U, phi, N*N*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_F, F, N*N*sizeof(double), hipMemcpyHostToDevice);
  
	//check for error
	cudaErrorCheck( hipPeekAtLastError() ); // Checks for launch error
	cudaErrorCheck( hipDeviceSynchronize() ); // Checks for execution error


	dim3 threadsPerBlock(16, 16);
	dim3 numBlocks(N / threadsPerBlock.x, N / threadsPerBlock.y);
  

	//loop for jacobi iterator
	//run until convergence set by errortol
	int jac = 0;
	double maxer = 1000.;
	double maxer2 = 100.;
	
	int wait = 0.;
	while( maxer > errortol){
		jac++;
	   

		//run twice with the pointers swapped to avoid having to copy memory 
 
		jacobi_iterator_GPU<<<numBlocks, threadsPerBlock>>>( d_U_new, d_U, d_F ,0);
		hipDeviceSynchronize();
		jacobi_iterator_GPU<<<numBlocks, threadsPerBlock>>>( d_U, d_U_new, d_F ,1);
		hipDeviceSynchronize();


		jacobi_iterator_GPU<<<numBlocks, threadsPerBlock>>>( d_U_new, d_U, d_F ,4);
		hipDeviceSynchronize();
		jacobi_iterator_GPU<<<numBlocks, threadsPerBlock>>>( d_U, d_U_new, d_F ,4);
		hipDeviceSynchronize();

   		//every 20 iteration check if the convergence condition has been met
    		if ((jac%1000) == 0){
			maxer2 = maxer;
	
			hipMemcpy(U_new, d_U_new, N*N*sizeof(double), hipMemcpyDeviceToHost);
			hipMemcpy(U, d_U, N*N*sizeof(double), hipMemcpyDeviceToHost);

    			hipDeviceSynchronize();

			
			for (int i = 0;i < N; i++){
				for (int j = 0; j < N;j++){
//					printf("%d %d %f\n",i,j,U[i*N+j]);
				}
			}

        		cudaErrorCheck( hipDeviceSynchronize() ); // Checks for execution error
			maxer = getmaxerror(U,U_new);

			printf("maximm error %.6E\n",maxer);
			if((maxer == maxer2) ){
				wait++;
				if (wait > 100.){
					break;
				}
			}
		}


	}

  	//check for errors with the kernal
  	cudaErrorCheck( hipPeekAtLastError() ); // Checks for launch error
  	cudaErrorCheck( hipDeviceSynchronize() ); // Checks for execution error

	//send the memory back to the cpu
//  	hipMemcpy(phi, d_U_new, N*N*sizeof(double), hipMemcpyDeviceToHost);
  	hipMemcpy(phi, d_U, N*N*sizeof(double), hipMemcpyDeviceToHost);


  	//third step: update the velocity
	for (int ix = 1;ix < N-1;ix++){
		for (int iy = 1; iy < N-1;iy++){
			ux[ix*N+iy] = uintx[ix*N+iy] - (dt/rho)*(phi[(ix+1)*N+iy] - phi[(ix-1)*N+iy])/(2*dx);
			uy[ix*N+iy] = uinty[ix*N+iy] - (dt/rho)*(phi[ix*N+iy+1] - phi[ix*N+iy-1])/(2*dx);
			printf("ix: %d iy: %d ux: %.5e uy: %.5e phi: %.5e\n",ix,iy,ux[ix*N+iy],uy[ix*N+iy],phi[ix*N+iy]);
		}
	}		
	
	for (int ix = 0;ix < N;ix++){
		for (int iy = 0; iy < N;iy++){



			//set normals to left and right to be zero
			if ( (ix == 0)){
				//uy[ix*N+iy] = uy[(ix+1)*N+iy];
				uy[ix*N+iy] = 0.;
				ux[ix*N+iy] = 0.;
			}
			if ( (ix == (N-1))){
				uy[ix*N+iy] = 0.;
				//uy[ix*N+iy] = uy[(ix-1)*N+iy] ;
				ux[ix*N+iy] = 0.;
			}
			//boundary condition for inlet
			if (iy ==(0)){

				//uinty[(N)*N+iy] = uinty[(N-1)*N+iy];
				uy[N*ix+iy] = -u0;
				ux[N*ix+iy] = 0.;
			}
			//boundary condition for the outlet 
			if(iy == (N-1)){
				uy[N*ix+iy] = uy[N*ix+iy-1];
				ux[N*ix+iy] = 0.;	
			
			}


/*
			//boundary conditions for u_intermediate
			if ( (ix == 0) | (iy == 0) | (ix == (N-1)) | (iy == (N-1))){
				uy[ix*N+iy] = -u0;
				ux[ix*N+iy] = 0.;
			}
			//boundary condition for outlet
			if (ix ==(N-1)){

		//uinty[(N)*N+iy] = uinty[(N-1)*N+iy];
				uy[(ix)*N+iy] = -u0;
			}
			//boundary condition for the inlet
			if(ix == 0){
				uy[iy] = -u0;
			}*/
		}
	}



  	//fouth step: update the pressure

	for (int ix = 0; ix < N;ix++){
		for (int iy = 0; iy < N;iy++){
			p[ix*N+iy] = phi[ix*N+iy] + beta*p[ix*N+iy];
		}
	}

	//calulate divu to check that its small
	for (int ix = 1; ix < (N-1);ix++){
		for (int iy = 1; iy < (N-1);iy++){
			divu[ix*N+iy] = ux[(ix+1)*N+iy]-ux[(ix-1)*N+iy]+uy[ix*N+iy+1]-uy[ix*N+iy-1];
		}
	}



	printf("Iteration: %i\n",it);
	
	savemat(ux,"ux",it,N);
	savemat(uy,"uy",it,N);
	savemat(phi,"phi",it,N);
	savemat(F,"F",it,N);

	savemat(divu,"divu",it,N);

	  }// end time iteration



	free(U);
	free(U_new);
	free(F);
	hipFree(d_U);
	hipFree(d_U_new);
	hipFree(d_F);

}





